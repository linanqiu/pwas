/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>

using namespace std;

int patient_count;
int icd_count;
int* patients_host;
int* icds_host;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

void free_all() {
  CUDA_CHECK_RETURN(hipHostFree(patients_host));
  CUDA_CHECK_RETURN(hipHostFree(icds_host));
}

void read_patients() {
  patient_count = 0;
  string line;
  ifstream file("csv_data/patients_sorted_short.csv");

  if (file.is_open()) {
    // skip first line
    getline(file, line);

    while (getline(file, line)) {
      patient_count ++;
    }
  }

  printf("%d patients read\n", patient_count);
  CUDA_CHECK_RETURN(hipHostAlloc((void**) &patients_host, patient_count * 4 * sizeof(int), hipHostMallocDefault));
  patient_count = 0;
  file.clear();
  file.seekg(0, ios::beg);

  if (file.is_open()) {
    // skip first line
    getline(file, line);

    while (getline(file, line)) {
      char * dup = strdup(line.c_str());
      char * record = strtok(dup, ",");

      for (int i = 0; i < 4; i++) {
        patients_host[patient_count * 4 + i] = atoi(record);
        record = strtok(NULL, ",");
      }

      patient_count++;
    }
  }
}

void read_icds() {
  icd_count = 0;
  string line;
  ifstream file("csv_data/icds.csv");

  if (file.is_open()) {
    // skip first line
    getline(file, line);

    while (getline(file, line)) {
      icd_count ++;
    }
  }

  printf("%d ICDs read\n", icd_count);
  CUDA_CHECK_RETURN(hipHostAlloc((void**) &icds_host, icd_count * sizeof(int), hipHostMallocDefault));
  icd_count = 0;
  file.clear();
  file.seekg(0, ios::beg);

  if (file.is_open()) {
    // skip first line
    getline(file, line);

    while (getline(file, line)) {
      char * dup = strdup(line.c_str());
      char * record = strtok(dup, ",");
      icds_host[icd_count] = atoi(record);
      icd_count++;
    }
  }
}

int main(int argc, char* argv[]) {
  read_patients();
  read_icds();
  free_all();
}
